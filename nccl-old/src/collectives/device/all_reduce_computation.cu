/*************************************************************************
 * Copyright (c) 2015-2019, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "all_reduce_computation.h"
#include "common.h"
#include "collectives.h"

IMPL_COLL_R(ncclAllReduceComputation, ncclCollAllReduceComputation);
